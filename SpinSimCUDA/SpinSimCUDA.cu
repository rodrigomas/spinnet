#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cutil.h>		// timers

#include "stdafx.h"

BOOL APIENTRY DllMain( HMODULE hModule,
                       DWORD  ul_reason_for_call,
                       LPVOID lpReserved
					 )
{
	switch (ul_reason_for_call)
	{
	case DLL_PROCESS_ATTACH:
	case DLL_THREAD_ATTACH:
	case DLL_THREAD_DETACH:
	case DLL_PROCESS_DETACH:
		break;
	}
	return TRUE;
}



// cuda kernel (internal)
__global__ void some_calculations(float *a, unsigned int N, unsigned int M)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; 
	if (idx < N)
	{
		// note1: no need for shared memory here
		// note2: global memory access is coalesced
		//        (no structs, float only used)

		// do computations M times on each thread
		// to extend processor time
		for(unsigned int i = 0; i < M; i++)
		{
			// some easy arithmetics		
			a[idx] = a[idx] * a[idx] * 0.1 - a[idx] - 10;
		}
	}
}

// internal variable (example, not really necessary here)
static volatile int PRINT_ERRORS = 1;	// true

// cuda helper function (internal)
int checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if(hipSuccess != err) 
    {
        if(PRINT_ERRORS)
			printf("Cuda error: %s: %s.\n", msg, hipGetErrorString(err));
        return err;
    }       
	return 0; // hipSuccess
}

// external variable example
extern "C" { float __declspec(dllexport) sExecutionTime = -1; }

// variable wrapper function
extern "C" float __declspec(dllexport) __stdcall GetExecutionTime()
{
	return sExecutionTime;
}

// cuda wrapper function
extern "C" int __declspec(dllexport) __stdcall SomeCalculationsCU
	(
	float *a_h,							// pointer to input array
	const unsigned int N,				// input array size
	const unsigned int M,				// kernel M parameter
	const int cuBlockSize = 512,		// kernel block size (max 512)
	const int showErrors = 1			// show CUDA errors in console window
	)
{
    int tmp = PRINT_ERRORS;
	PRINT_ERRORS = showErrors;

	float *a_d;							// pointer to device array
    size_t size = N * sizeof(float);
	int cuerr = 0;						// no errors
	unsigned int timer = 0;
    
    hipMalloc((void**)&a_d, size);		// allocate array on device    
	hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
    
    int n_blocks = N / cuBlockSize + (N % cuBlockSize == 0 ? 0 : 1);
    
	cutCreateTimer(&timer);			    // from cutil.h
	cutStartTimer(timer);
	some_calculations <<<n_blocks, cuBlockSize>>> (a_d, N, M);	// kernel invocation
	hipDeviceSynchronize();			// by default kernel runs in parallel with CPU code
	cutStopTimer(timer);
    
	cuerr = checkCUDAError("cuda kernel");

    hipMemcpy(a_h, a_d, size, hipMemcpyDeviceToHost);   
	if(!cuerr) cuerr = checkCUDAError("cuda memcpy");
	
	sExecutionTime = cutGetTimerValue(timer);
	
    hipFree(a_d);
    if(!cuerr) cuerr = checkCUDAError("cuda free");

	PRINT_ERRORS = tmp;
	return cuerr;
}

// cpu version for comparison
extern "C" void __declspec(dllexport) __stdcall SomeCalculationsCPU
	(
	float *a_h, 
	const unsigned int N,
	const unsigned int M
	)
{
	unsigned int timer = 0;
	cutCreateTimer(&timer);
	cutStartTimer(timer);
	for(unsigned int i = 0; i < N; i++)
		for(unsigned int j = 0; j < M; j++)
			*(a_h + i) = *(a_h + i) * *(a_h + i) * 0.1 - *(a_h + i) - 10;
	cutStopTimer(timer);
	sExecutionTime = cutGetTimerValue(timer);
}
