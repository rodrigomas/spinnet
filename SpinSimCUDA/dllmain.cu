#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>

#include "targetver.h"

#define WIN32_LEAN_AND_MEAN             // Exclude rarely-used stuff from Windows headers
// Windows Header Files:
#include <windows.h>

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
printf("Error at %s:%d\n",__FILE__,__LINE__); \
return EXIT_FAILURE;}} while(0)
#define CURAND_CALL(x) do { if((x) != HIPRAND_STATUS_SUCCESS) { \
printf("Error at %s:%d\n",__FILE__,__LINE__); \
return EXIT_FAILURE;}} while(0)

BOOL APIENTRY DllMain( HMODULE hModule,
                       DWORD  ul_reason_for_call,
                       LPVOID lpReserved
					 )
{
	switch (ul_reason_for_call)
	{
	case DLL_PROCESS_ATTACH:
	case DLL_THREAD_ATTACH:
	case DLL_THREAD_DETACH:
	case DLL_PROCESS_DETACH:
		break;
	}
	return TRUE;
}


__device__ void load_neightbours(int i, int j, int lines, int cols, int* m, int* neighbours)
{
    // UP
    if (i == 0) neighbours[0] = m[((lines - 1) * lines + j)];
    else neighbours[0] = m[((i - 1) * lines + j)];

    //LEFT
    if (j == cols - 1) neighbours[1] = m[(i * lines + 0)];
    else neighbours[1] = m[(i * lines + j + 1)];

    // Down
    if (i == lines - 1) neighbours[2] = m[(0 + j)];
    else neighbours[2] = m[((i + 1) * lines + j)];

    //RIGHT
    if (j == 0) neighbours[3] = m[(i * lines + cols - 1)];
    else neighbours[3] = m[(i * lines + j - 1)];
}



// cuda kernel (internal)
__global__ void spin_simulate(int *m, float *r, unsigned int N, int nAtoms, float J, float T)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x; 

	if (idx < N)
	{
		// note1: no need for shared memory here
		// note2: global memory access is coalesced
		//        (no structs, float only used)

		// do computations M times on each thread
		// to extend processor time
		//for(unsigned int i = 0; i < M; i++)
		//{
		//	// some easy arithmetics		
		//	a[idx] = a[idx] * a[idx] * 0.1 - a[idx] - 10;
		//}

		float pr = r[idx];

		int p = (int)(pr * N); //Rnd.Next(0, size);//Position[i * nAtoms + j];

		int ii = p / nAtoms;
		int jj = p % nAtoms;

		int neighbours[4] = {1,1,-1,-1};

		load_neightbours(ii, jj, nAtoms, nAtoms, m, neighbours);

		double dE = 0;
		double E0 = 0;
		double Ef = 0;

		int Sum = 0;

		for (int k = 0; k < 4; k++)
		{
			Sum += m[p] * neighbours[k];
		}

		E0 = -(J / T) * Sum;
		Ef = (J / T) * Sum;

		dE = Ef - E0;

		if (dE < 0)
		{
			m[p] = -m[p];
		}
		else
		{
			//double E = Math.Exp(-dE / (K * T));
			float E =  expf(-dE / (T));

			float Z = (pr + r[p]) /2.0f;//Rnd.NextDouble();

			if (E >= Z)
			{
				m[p] = -m[p];
			}                            
		}
	}
}

// internal variable (example, not really necessary here)
//static volatile int PRINT_ERRORS = 1;	// true

// cuda wrapper function
extern "C" int __declspec(dllexport) __stdcall SpinSimulateGPU
	(
	int *m,						
	int N,				
	int nAtoms,			
	float J, float T,
	int Steps,
	const int cuBlockSize = 512
	)
{
	int *m_d;							// pointer to device array
	float *d_r;							// pointer to device array
    size_t size = N * sizeof(int);

	hiprandGenerator_t gen;

	CURAND_CALL(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
	CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL));

	CUDA_CALL(hipMalloc((void **)&d_r, N * sizeof(float)));	
   
    CUDA_CALL(hipMalloc((void**)&m_d, size));		// allocate array on device    
	CUDA_CALL(hipMemcpy(m_d, m, size, hipMemcpyHostToDevice));
    
    int n_blocks = N / cuBlockSize + (N % cuBlockSize == 0 ? 0 : 1);

	for(int i = 0; i < Steps; i++)
	{
		CURAND_CALL(hiprandGenerateUniform(gen, d_r, N));

		spin_simulate <<<n_blocks, cuBlockSize>>> (m_d, d_r, N, nAtoms, J, T);	// kernel invocation
	}	

	CUDA_CALL(hipDeviceSynchronize());			// by default kernel runs in parallel with CPU code
    
    CUDA_CALL(hipMemcpy(m, m_d, size, hipMemcpyDeviceToHost));   	
	
	CURAND_CALL(hiprandDestroyGenerator(gen));

    CUDA_CALL(hipFree(m_d));

	CUDA_CALL(hipFree(d_r));

	return 0;
}